#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <locale.h>
#include <iostream>

int n = 3; // ����������� ������������
double a[] = { 0, 0, 0 }; // ������ ������� �������� ����������
double b[] = { 100, 100, 100 }; // ������� ������� �������� ����������
int R = 10000; // ���������� ��������
double e1 = 1e-7; // �������� ���������� 
double e2 = 1e-7; // �������� ���������� 
double t = 0.5; // �������� ����

double p[] = { 10, 20, 30 };

// ����������� �������
// ����� ������ ���� ������� �������� ������� s
__global__ void f(double *x, double *p, double *s)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	s[id] = (x[id] - p[id])*(x[id] - p[id]);
}

__global__ void gradientvector(double *x, double *p, double *g)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	g[id] = 2.0*(x[id] - p[id]);
}

// ������������� ���������� ��������������� �����
__global__ void setuprand(hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x; /* Each thread gets same seed, a different sequence number, no offset */
	hiprand_init(1234, id, 0, &state[id]);
}

// ��������� ���������������� �������
__global__ void randvector(double *x, double *a, double *b, hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState localState = state[id];
	double p = hiprand_uniform_double(&localState);
	state[id] = localState;
	x[id] = a[id] + p*(b[id] - a[id]);
}

// ��������� ���������������� �������
__global__ void nextvector(double *x1, double *x, double *g, double *a, double *b, double t)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	x1[id] = x[id] - t*g[id];
	if (x1[id] < a[id]) x1[id] = a[id];
	if (x1[id] > b[id]) x1[id] = b[id];
}


int main()
{
	// ��������� ��������� � ������� Windows
	// ������� setlocale() ����� ��� ���������, ������ �������� - ��� ��������� ������, � ����� ������ LC_TYPE - ����� ��������, ������ �������� � �������� ������. 
	// ������ ������� ��������� ����� ������ "Russian", ��� ��������� ������ ������� �������, ����� ����� �������� ����� ����� �� ��� � � ��.
	setlocale(LC_ALL, "");

	hiprandState *devStates;
	hipMalloc((void **)&devStates, n*sizeof(hiprandState));
	setuprand << <1, n >> >(devStates);

	double *devA, *devB, *devX, *devX1, *devP, *devS, *devG;
	double *x, *x1, *s, *g;
	hipMalloc((void **)&devA, n*sizeof(double));
	hipMalloc((void **)&devB, n*sizeof(double));
	hipMalloc((void **)&devX, n*sizeof(double));
	hipMalloc((void **)&devX1, n*sizeof(double));
	hipMalloc((void **)&devP, n*sizeof(double));
	hipMalloc((void **)&devS, n*sizeof(double));
	hipMalloc((void **)&devG, n*sizeof(double));
	x = (double *)malloc(n*sizeof(double));
	x1 = (double *)malloc(n*sizeof(double));
	s = (double *)malloc(n*sizeof(double));
	g = (double *)malloc(n*sizeof(double));

	hipMemcpy(devA, a, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devB, b, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devP, p, n*sizeof(double), hipMemcpyHostToDevice);

	std::cout << "����� ��������� �����" << std::endl;

	randvector <<<1, n>>>(devX, devA, devB, devStates);
	f <<<1, n>>>(devX, devP, devS);
	hipMemcpy(s, devS, n*sizeof(double), hipMemcpyDeviceToHost);
	double fx = 0; for (int i = 0; i < n; i++) fx += s[i];

	for (;;)
	{
		gradientvector<<<1,n>>>(devX,devP,devG);
		double d = 0;
		for (auto i = 0; i < n; i++) d += g[i] * g[i];
		d = sqrt(d);
		if (d < e1) break;

		auto fx1 = fx;

		std::cout << "����� ��������� �����" << std::endl;

		double l = 0;
		for (auto t1 = t;; t1 /= 2)
		{
			nextvector <<<1, n>>>(devX1, devX, devG, devA, devB, t1);

			f <<<1, n>>>(devX1, devP, devS);
			hipMemcpy(s, devS, n*sizeof(double), hipMemcpyDeviceToHost);
			fx1 = 0; for (int i = 0; i < n; i++) fx1 += s[i];

			hipMemcpy(x, devX, n*sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(x1, devX1, n*sizeof(double), hipMemcpyDeviceToHost);
			l = 0;
			for (auto i = 0; i < n; i++) l += (x[i] - x1[i]) * (x[i] - x1[i]);
			l = sqrt(l);
			if (fx1 < fx) break;
			if (l < e2) break;
		}
		if (l < e2 && abs(fx - fx1) < e2)
		{
			hipMemcpy(devX, devX1, n*sizeof(double), hipMemcpyDeviceToDevice);
			fx = fx1;
			break;
		}
		hipMemcpy(devX, devX1, n*sizeof(double), hipMemcpyDeviceToDevice);
		fx = fx1;
	}
	// ����� �����������
	hipMemcpy(x, devX, n*sizeof(double), hipMemcpyDeviceToHost);

	std::cout << "����� �������� : ";
	for (auto i = 0; i < n; i++)
	{
		std::cout << x[i];
		if (i < n - 1) std::cout << ",";
	}
	std::cout << std::endl;

	std::cout << "�������� �������� : " << fx << std::endl;

	free(x);
	free(x1);
	free(s);
	free(g);
	hipFree(devX);
	hipFree(devX1);
	hipFree(devA);
	hipFree(devB);
	hipFree(devP);
	hipFree(devS);
	hipFree(devG);
	hipFree(devStates);

	getchar(); // �������� ����� � ���������� ����� ����������� ���������
	return 0;
}

