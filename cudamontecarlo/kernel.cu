
#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>

#include <locale.h>
#include <iostream>

int n = 3; // ����������� ������������
double a[] = { 0, 0, 0 }; // ������ ������� �������� ����������
double b[] = { 100, 100, 100 }; // ������� ������� �������� ����������
int R = 10000; // ���������� ��������
double e = 1e-7; // �������� ���������� 

double p[] = { 10, 20, 30 };

// ����������� �������
// ����� ������ ���� ������� �������� ������� s
__global__ void f(double *x, double *p, double *s)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	s[id] = (x[id] - p[id])*(x[id] - p[id]);
}

// ������������� ���������� ��������������� �����
__global__ void setuprand(hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x; /* Each thread gets same seed, a different sequence number, no offset */
	hiprand_init(1234, id, 0, &state[id]);
}

// ��������� ���������������� �������
__global__ void randvector(double *x, double *a, double *b, hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState localState = state[id]; 
	double p = hiprand_uniform_double(&localState);
	state[id] = localState; 
	x[id] = a[id] + p*(b[id] - a[id]);
}

int main(int argc, char* argv[])
{
	// ��������� ��������� � ������� Windows
	// ������� setlocale() ����� ��� ���������, ������ �������� - ��� ��������� ������, � ����� ������ LC_TYPE - ����� ��������, ������ �������� � �������� ������. 
	// ������ ������� ��������� ����� ������ "Russian", ��� ��������� ������ ������� �������, ����� ����� �������� ����� ����� �� ��� � � ��.
	setlocale(LC_ALL, "");

	hiprandState *devStates;
	hipMalloc((void **)&devStates, n*sizeof(hiprandState));
	setuprand <<<1, n>>>(devStates);

	double *devA, *devB, *devX, *devX1, *devP, *devS;
	double *x, *s;
	hipMalloc((void **)&devA, n*sizeof(double));
	hipMalloc((void **)&devB, n*sizeof(double));
	hipMalloc((void **)&devX, n*sizeof(double));
	hipMalloc((void **)&devX1, n*sizeof(double));
	hipMalloc((void **)&devP, n*sizeof(double));
	hipMalloc((void **)&devS, n*sizeof(double));
	x = (double *)malloc(n*sizeof(double));
	s = (double *)malloc(n*sizeof(double));

	hipMemcpy(devA, a, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devB, b, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devP, p, n*sizeof(double), hipMemcpyHostToDevice);

	// ����� ��������� �����
	randvector <<<1, n>>>(devX, devA, devB, devStates);
	f <<<1, n>>>(devX, devP, devS);
	hipMemcpy(s, devS, n*sizeof(double), hipMemcpyDeviceToHost);
	double fx = 0; for (int i = 0; i < n; i++) fx += s[i];

	for (auto r = 0; r < R; r++)
	{
		// ����� ��������� �����
		randvector <<<1, n>>>(devX1, devA, devB, devStates);
		f <<<1, n>>>(devX1, devP, devS);
		hipMemcpy(s, devS, n*sizeof(double), hipMemcpyDeviceToHost);
		double fx1 = 0; for (int i = 0; i < n; i++) fx1 += s[i];

		if (fx < fx1) continue;
		if (abs(fx - fx1)<e)
		{
			hipMemcpy(devX, devX1, n*sizeof(double), hipMemcpyDeviceToDevice);
			fx = fx1;
			break;
		}
		hipMemcpy(devX, devX1, n*sizeof(double), hipMemcpyDeviceToDevice);
		fx = fx1;
	}

	// ����� �����������
	hipMemcpy(x, devX, n*sizeof(double), hipMemcpyDeviceToHost);

	std::cout << "����� �������� : ";
	for (auto i = 0; i < n; i++)
	{
		std::cout << x[i];
		if (i < n - 1) std::cout << ",";
	}
	std::cout << std::endl;

	std::cout << "�������� �������� : " << fx << std::endl;

	free(x);
	free(s);
	hipFree(devX);
	hipFree(devX1);
	hipFree(devA);
	hipFree(devB);
	hipFree(devP);
	hipFree(devS);
	hipFree(devStates);

	getchar(); // �������� ����� � ���������� ����� ����������� ���������
	return 0;
}

